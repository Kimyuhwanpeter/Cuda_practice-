#include <../local/cuda-11.0/targets/x86_64-linux/include/hip/hip_runtime.h>
#include <../local/cuda-11.0/targets/x86_64-linux/include/hip/hip_runtime.h>
#include <../local/cuda-11.0/targets/x86_64-linux/include/hip/hip_runtime_api.h>
#include <../include/hipDNN.h>
#include <../local/cuda-11.0/targets/x86_64-linux/include/>

#include <iostream>
#include <stdio.h>

#include "/yuhwan/yuhwan/opencv2/world.hpp"
#include "/yuhwan/yuhwan/opencv2/core.hpp"
#include "/yuhwan/yuhwan/opencv2/highgui.hpp"
#include "/yuhwan/yuhwan/opencv2/imgproc.hpp"
#include "/yuhwan/yuhwan/opencv2/core/mat.hpp"

// /usr/bin/g++ -fdiagnostics-color=always -g /yuhwan/yuhwan/Projects/CUDA/parctice/test.cpp -o /yuhwan/yuhwan/Projects/CUDA/parctice/test -lstdc++ -I/usr/local/cuda-11.0/include -L/usr/local/cuda-11.0/lib64 -Icudart -Icuda -std=c++11

// using namespace std;

int main(int argc, char** argv)
{
    
    cv::Mat();
    
    hipDeviceProp_t prob;
    
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    std::cout << "Found " << numGPUs << " GPUs." << std::endl;
    hipSetDevice(0);

    for(int i = 0; i < numGPUs; i++)
    {
        hipGetDeviceProperties(&prob, i);
        std::cout << "Device name: " << prob.name << std::endl;
        std::cout << "Global ram: " << prob.totalGlobalMem << std::endl;
        std::cout << "Constent ram: " << prob.totalConstMem << std::endl;
    }
    
    hipdnnDataType_t dtype = HIPDNN_DATA_FLOAT;
    hipdnnTensorFormat_t format = HIPDNN_TENSOR_NHWC;

    hipdnnHandle_t handle_;
    hipdnnCreate(&handle_);

    // Define descroptor
    int b = 1, c = 1, h = 1, w = 10;
    int NUM_ELEMENTS = b*h*w*c;
    hipdnnTensorDescriptor_t x_desc;
    hipdnnCreateTensorDescriptor(&x_desc);
    hipdnnSetTensor4dDescriptor(x_desc, format, dtype, b, h, w, c);
    
    // Create tensor
    float *x;
    hipMallocManaged(&x, NUM_ELEMENTS * sizeof(float));
    for(int i=0; i<NUM_ELEMENTS; i++)
    {
        x[i] = i * 1.00f;
    }
    std::cout << "Original array: ";
    for(int i=0; i<NUM_ELEMENTS; i++) std::cout << x[i] << ", ";

    // create activation function descriptor
    float alpha[1] = {1};
    float beta[1] = {0.0};
    hipdnnActivationDescriptor_t sigmoid_activation;
    hipdnnActivationMode_t mode = HIPDNN_ACTIVATION_SIGMOID;
    hipdnnNanPropagation_t prop = HIPDNN_NOT_PROPAGATE_NAN;
    hipdnnCreateActivationDescriptor(&sigmoid_activation);
    hipdnnSetActivationDescriptor(sigmoid_activation, mode, prop, 0.0f);

    hipdnnActivationForward(
        handle_,
        sigmoid_activation,
        alpha,
        x_desc,
        x,
        beta,
        x_desc,
        x
    );
    

    hipdnnDestroy(handle_);
    std::cout << std::endl << "Destroyed cuDNN handle." << std::endl;
    std::cout << "New array: ";
    for(int i=0;i<NUM_ELEMENTS;i++) std::cout << x[i] << " ";
    std::cout << std::endl;
    hipFree(x);
    return 0;

}
